#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

__global__
void find_max(const float* const d_logLuminance,
		       const size_t numPixels,
		       float *d_max_logLum
		       )
{
  extern __shared__ float sdata[];

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int tid  = threadIdx.x;

  // load shared mem from global mem
  sdata[tid] = d_logLuminance[x];
  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      if(sdata[tid]<sdata[tid+s]) {
	sdata[tid] = sdata[tid + s];
      }
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0)
  {
    d_max_logLum[blockIdx.x] = sdata[0];
  }
}

__global__
void find_min(const float* const d_logLuminance,
		       const size_t numPixels,
		       float *d_min_logLum
		       )
{
  extern __shared__ float sdata[];

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int tid  = threadIdx.x;

  // load shared mem from global mem
  sdata[tid] = d_logLuminance[x];
  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      if(sdata[tid]>sdata[tid+s]) {
	sdata[tid] = sdata[tid + s];
      }
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0)
  {
    d_min_logLum[blockIdx.x] = sdata[0];
  }
}

__global__
void gen_histo(const float* const d_logLuminance,
		       const size_t numPixels,
		       const size_t numBins,
		       unsigned int *d_histo,
		       const float logLumMin,
		       const float logLumRange
		       )
{

  extern __shared__  int s_histo[];

  int x = blockIdx.x * blockDim.x + threadIdx.x;

  for(int idx=threadIdx.x;idx<numBins;idx+=blockDim.x) {
    s_histo[idx] = 0;
  }
  __syncthreads();            // make sure entire block is loaded!

  if (x<numPixels) {
    unsigned int bin = static_cast<unsigned int>((d_logLuminance[x] - logLumMin) / logLumRange * numBins);
    if(bin>=numBins) {
      bin=numBins-1;
    }
    atomicAdd(&(s_histo[bin]), 1);
  }
  __syncthreads();            // make sure entire block is loaded!

  for(int idx=threadIdx.x;idx<numBins;idx+=blockDim.x) {
      atomicAdd(&(d_histo[idx]), s_histo[idx]);
  }
}

__global__
void get_cdf(const unsigned int * const d_histo,
		       const size_t numBins,
		       unsigned int * d_cdf
		       )
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if(x!=0) {
    return;
  }

  d_cdf[0]=0;
  for(int i=1;i<numBins;i++) {
    d_cdf[i]=d_histo[i-1]+d_cdf[i-1];
  }
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  //Step 1
  //first we find the minimum and maximum across the entire image

  float *d_min_logLum;
  float *d_max_logLum;
  size_t numPixels=numCols*numRows;
  {
    const dim3 blockSize(256, 1, 1);
    dim3 gridSize((numPixels+blockSize.x-1)/blockSize.x , 1 , 1);

    checkCudaErrors(hipMalloc(&d_min_logLum,gridSize.x*sizeof(float)));

    find_min<<<gridSize, blockSize, blockSize.x*sizeof(float)>>>(d_logLuminance, numPixels, d_min_logLum);

    checkCudaErrors(hipMalloc(&d_max_logLum,gridSize.x*sizeof(float)));
    find_max<<<gridSize, blockSize, blockSize.x*sizeof(float)>>>(d_logLuminance, numPixels, d_max_logLum);

    while(gridSize.x>1) {
      int groupSize=gridSize.x;
      gridSize.x=(groupSize+blockSize.x-1)/blockSize.x;
      find_min<<<gridSize, blockSize, blockSize.x*sizeof(float)>>>(d_min_logLum, groupSize, d_min_logLum);
      find_max<<<gridSize, blockSize, blockSize.x*sizeof(float)>>>(d_max_logLum, groupSize, d_max_logLum);
    }
  }

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(&min_logLum, d_min_logLum, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_max_logLum, sizeof(float), hipMemcpyDeviceToHost));

  //Step 2 && Step 3
  unsigned int *d_histo;
  {
    const dim3 blockSize(256, 1, 1);
    const dim3 gridSize((numPixels+blockSize.x-1)/blockSize.x , 1 , 1);

    checkCudaErrors(hipMalloc(&d_histo,numBins*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(d_histo,0,numBins*sizeof(unsigned int)));

    gen_histo<<<gridSize, blockSize,numBins*sizeof(unsigned int)>>>(d_logLuminance, numPixels, numBins,d_histo,min_logLum,max_logLum-min_logLum);

    // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
    // launching your kernel to make sure that you didn't make any mistakes.
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  }

  //Step 4
  {
    const dim3 blockSize(256, 1, 1);
    get_cdf<<<1, blockSize>>>(d_histo, numBins,d_cdf);

    // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
    // launching your kernel to make sure that you didn't make any mistakes.
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  }
}
